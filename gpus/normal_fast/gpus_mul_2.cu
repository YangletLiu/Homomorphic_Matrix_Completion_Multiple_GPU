#include "hip/hip_runtime.h"
/*************************************************************************
	> File Name: trust.cpp
	> Author: hanlu
	> Mail: hanlu@shu.edu.cn 
	> Created Time: 2018年12月10日 星期一 21时13分39秒
 ************************************************************************/

#include "head.h"

int main(int argc,char *argv[]){

int luhan[11]={1000,222,500,1000,1500,2000,2500,3000,3500,4000,4500};
for(int hh=0;hh<11;hh++){
	int M = luhan[hh];
	int N = M;
	int R = int(0.01*M);
    
	int count_device = 0;
    	hipGetDeviceCount(&count_device);
    	if(count_device == 0){

    		printf("There is no GPUs\n");
	    	return false;
    	}
    	printf("There are %d GPUs\n",count_device);
    	for(int i=0;i<count_device;i++){
		hipDeviceProp_t prop;
	    	if(hipGetDeviceProperties(&prop,i) == hipSuccess){
			printf("GPU %d: \t %s \n",i,prop.name);
	    	}
    	}
	int gpu_0 = 0;
	int gpu_1 = 1;

	srand((unsigned)time(NULL));
	dt *A = new dt[M*R]();
	for(int i = 0;i<M*R;i++){
		A[i] = rand()*0.1/(RAND_MAX*0.1);
	}
	dt *B = new dt[N*R]();
	for(int i = 0;i<N*R;i++){
		B[i] = rand()*0.1/(RAND_MAX*0.1);
	}

	dt *D = new dt[M*N]();
	matrixProduct(A,B,D,M,R,N); // D is the origin data
//	cout<<"origin data"<<endl;
//	printTensor(D,1,M*N,1);   // we view it store as col
	delete[] A;A=nullptr;
	delete[] B;B=nullptr;

// get transpose of D  N*M
	dt *D1 = new dt[M*N]();
	for(int i = 0;i<M;i++){
		for(int j = 0;j<N;j++){
			D1[i*N+j] = D[j*M+i];
		}
	}
//	printTensor(D1,1,M*N,1);

//	dt Omega[M*N]={1,0,1,1,0,1,1,0,1,1,0,1};
	dt *Omega = new dt[M*N]();
	for(int i = 0;i<M*N;i++){
		Omega[i] = rand()*0.1/(RAND_MAX*0.1);
		if(Omega[i]<=0.1){
			Omega[i] = 1;
		}else{
			Omega[i] = 0;
		}
	}
	cout<<"sample is 0.5"<<endl;

//	printTensor(Omega,1,M*N,1);
	dt *Omega1 = new dt[M*N]();
	for(int i = 0;i<M;i++){
		for(int j = 0;j<N;j++){
			Omega1[i*N+j] = Omega[j*M+i];
		}
	}
//	printTensor(Omega1,1,M*N,1);

//	dt U[M*R] = {3,6,2,6,2,6,1,8};
	dt *U = new dt[M*R]();
	for(int i = 0;i<M*R;i++){
		U[i] = rand()*0.1/(RAND_MAX*0.1);
	}

	dt *V = new dt[R*N]();

	dt *PB = new dt[M*K]();   //public data size of M*R store as column
	for(int i = 0;i<M*K;i++){
		PB[i] = D[i];
	}
	
	dt *MM = new dt[M*N]();  //MM is encpypted matrix
	dt *MM1 = new dt[M*N]();  //MM1 is its transpose

	Enc(D,Omega,PB,MM,MM1,M,N);
/* MM is incom encryped matrix of size m*n
   Omega is sample matrix
   we divide n into two part and each solve on one gpu   
*/
	int pl = N/2;    // divide MM M*N into pl and pr
	int pr = N-N/2;
	int ptl = M/2;   // divide MM1 N*M into ptl and ptr
	int ptr = M-M/2;
    
	double start, end;
    	start = omp_get_wtime();
// malloc  dta part to device 0
	hipSetDevice(gpu_0);
	dt *dp_MM,*dp_MM1,*dp_U,*dp_V,*dp_Omega,*dp_Omega1;
	dt *dp_left,*dp_right,*dp_left1,*dp_right1;
	hipMalloc((void**)&dp_MM,sizeof(dt)*M*pl);
	hipMalloc((void**)&dp_MM1,sizeof(dt)*N*ptl);
	hipMalloc((void**)&dp_U,sizeof(dt)*M*R);
	hipMalloc((void**)&dp_V,sizeof(dt)*R*N);
	hipMalloc((void**)&dp_Omega,sizeof(dt)*M*pl);
	hipMalloc((void**)&dp_Omega1,sizeof(dt)*N*ptl);
	hipMalloc((void**)&dp_left,sizeof(dt)*R*R*pl);
	hipMalloc((void**)&dp_right,sizeof(dt)*R*1*pl);
	hipMalloc((void**)&dp_left1,sizeof(dt)*R*R*ptl);
	hipMalloc((void**)&dp_right1,sizeof(dt)*R*1*ptl);

// malloc  dta part to device 1
	hipSetDevice(gpu_1);
	dt *dq_MM,*dq_MM1,*dq_U,*dq_V,*dq_Omega,*dq_Omega1;
	dt *dq_left,*dq_right,*dq_left1,*dq_right1;
	hipMalloc((void**)&dq_MM,sizeof(dt)*M*pr);
	hipMalloc((void**)&dq_MM1,sizeof(dt)*N*ptr);
	hipMalloc((void**)&dq_U,sizeof(dt)*M*R);
	hipMalloc((void**)&dq_V,sizeof(dt)*R*N);
	hipMalloc((void**)&dq_Omega,sizeof(dt)*M*pr);
	hipMalloc((void**)&dq_Omega1,sizeof(dt)*N*ptr);
	hipMalloc((void**)&dq_left,sizeof(dt)*R*R*pr);
	hipMalloc((void**)&dq_right,sizeof(dt)*R*1*pr);
	hipMalloc((void**)&dq_left1,sizeof(dt)*R*R*ptr);
	hipMalloc((void**)&dq_right1,sizeof(dt)*R*1*ptr);

// transfer data to GPU
	hipSetDevice(gpu_0);
	hipMemcpy(dp_MM,MM,sizeof(dt)*M*pl,hipMemcpyHostToDevice);
	hipMemcpy(dp_MM1,MM1,sizeof(dt)*N*ptl,hipMemcpyHostToDevice);
	hipMemcpy(dp_U,U,sizeof(dt)*M*R,hipMemcpyHostToDevice);
	hipMemcpy(dp_V,V,sizeof(dt)*R*N,hipMemcpyHostToDevice);
	hipMemcpy(dp_Omega,Omega,sizeof(dt)*M*pl,hipMemcpyHostToDevice);
	hipMemcpy(dp_Omega1,Omega1,sizeof(dt)*N*ptl,hipMemcpyHostToDevice);
// transfer data to GPU
	hipSetDevice(gpu_1);
	hipMemcpy(dq_MM,MM+M*pl,sizeof(dt)*M*pr,hipMemcpyHostToDevice);
	hipMemcpy(dq_MM1,MM1+N*ptl,sizeof(dt)*N*ptr,hipMemcpyHostToDevice);
	hipMemcpy(dq_U,U,sizeof(dt)*M*R,hipMemcpyHostToDevice);
	hipMemcpy(dq_V,V,sizeof(dt)*R*N,hipMemcpyHostToDevice);
	hipMemcpy(dq_Omega,Omega+M*pl,sizeof(dt)*M*pr,hipMemcpyHostToDevice);
	hipMemcpy(dq_Omega1,Omega1+N*ptl,sizeof(dt)*N*ptr,hipMemcpyHostToDevice);

// compute each part
	for(int i = 0;i<10;i++){
	
		hipSetDevice(gpu_0);
		getbatch(dp_U,dp_MM,dp_Omega,dp_left,dp_right,M,pl,R);
		lsq(dp_left,dp_right,dp_V,M,pl,R);
		hipMemcpyPeerAsync(dq_V,gpu_1,dp_V,gpu_0,sizeof(dt)*R*pl);

		hipSetDevice(gpu_1);
		getbatch(dq_U,dq_MM,dq_Omega,dq_left,dq_right,M,pr,R);
		lsq(dq_left,dq_right,dq_V+R*pl,M,pr,R);
		hipMemcpyPeerAsync(dp_V+R*pl,gpu_0,dq_V+R*pl,gpu_1,sizeof(dt)*R*pr);
		hipDeviceSynchronize();

		hipSetDevice(gpu_0);
		Matrixtranpose(dp_V,gpu_0,N,R);
		hipSetDevice(gpu_1);
		Matrixtranpose(dq_V,gpu_1,N,R);
		hipDeviceSynchronize();
		
		hipSetDevice(gpu_0);
		getbatch(dp_V,dp_MM1,dp_Omega1,dp_left1,dp_right1,N,ptl,R);
		lsq(dp_left1,dp_right1,dp_U,N,ptl,R);
		hipMemcpyPeerAsync(dq_U,gpu_1,dp_U,gpu_0,sizeof(dt)*R*ptl);

		hipSetDevice(gpu_1);
		getbatch(dq_V,dq_MM1,dq_Omega1,dq_left1,dq_right1,N,ptr,R);
		lsq(dq_left1,dq_right1,dq_U+R*ptl,N,ptr,R);
		hipMemcpyPeerAsync(dp_U+R*ptl,gpu_0,dq_U+R*ptl,gpu_1,sizeof(dt)*R*ptr);
		hipDeviceSynchronize();

		hipSetDevice(gpu_0);
		Matrixtranpose(dp_U,gpu_0,M,R);
		hipSetDevice(gpu_1);
		Matrixtranpose(dq_U,gpu_1,M,R);
		hipDeviceSynchronize();

	}
	// we need to U*V
	dt *result1 = new dt[M*N]();
	mproduct(dq_U,dq_V,result1,M,R,N);
	hipDeviceSynchronize();

	hipSetDevice(gpu_0);
	hipFree(dp_MM); hipFree(dp_MM1); hipFree(dp_U);
	hipFree(dp_V); hipFree(dp_Omega); hipFree(dp_Omega1);
	hipFree(dp_left); hipFree(dp_right); hipFree(dp_left1);
	hipFree(dp_right1);
    
	hipSetDevice(gpu_1);
	hipFree(dq_MM); hipFree(dq_MM1); hipFree(dq_U);
	hipFree(dq_V); hipFree(dq_Omega); hipFree(dq_Omega1);
	hipFree(dq_left); hipFree(dq_right); hipFree(dq_left1);
	hipFree(dq_right1);

    	end = omp_get_wtime();
    	double time = end-start;
    	cout<<time<<endl;

	dt *result = new dt[M*N]();
	Dec(result1,result,PB,M,N); // decrypt result1 to result

	delete[] Omega1;Omega1=nullptr;
	delete[] Omega;Omega=nullptr;
	delete[] D1;D1=nullptr;
	delete[] U;U=nullptr;
	delete[] V;V=nullptr;
//	printTensor(result,1,M*N,1);
	

	double sh = 0.0;
	double xia = 0.0;
	for(int i = 0;i<M*N;i++){
		sh+=(result[i]-D[i])*(result[i]-D[i]);
		xia+=(D[i]*D[i]);
	}
	delete[] result;result=nullptr;
	delete[] result1;result1=nullptr;
	delete[] D;D=nullptr;
	delete[] MM;MM=nullptr;
	delete[] MM1;MM1=nullptr;
	double error=0.0;
	error = sqrt(sh)/sqrt(xia);
	cout<<"error is  "<<error<<endl;
	ofstream outfile("v100q6000.txt",ios::app);
	outfile<<M<<"*"<<N<<" "<<R<<" ";
//	outfile<<(double)(t2-t1)/CLOCKS_PER_SEC<<"s"<<"  ";
	outfile<<time<<"s"<<"  ";
	outfile<< error<<endl;
	outfile.close();
}
	return 0;
}

