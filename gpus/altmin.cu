#include "head.h"

void lsq(dt *left,dt *right,dt *res,int m,int n,int r){
	//left is r*r*n res is r*1*n
	//   r*r*batch      r*nrhs*batch
	// right  is r*1*n
	magma_init();
	magma_queue_t queue = NULL;
	magma_int_t dev = 0;
	magma_getdevice(&dev);
	magma_queue_create(dev,&queue);
	
	dt *d_left, *d_right; //origin data

	magma_int_t *d_ipiv;
	magma_int_t **dipiv_array;

	magma_int_t row, nrhs, lddl, lddr,batch;

	magmaFloat_ptr *darray_left;
	magmaFloat_ptr *darray_right;  //transfered

	magma_int_t *dinfo_array;

	row = r;
	nrhs = 1;
	batch = n;
	
	lddl = magma_roundup(row,32);
	lddr = magma_roundup(row,32);
	
	magma_smalloc(&d_left,lddl*r*batch);
	magma_smalloc(&d_right,lddr*1*batch);
	magma_imalloc(&d_ipiv,r*batch);
	magma_imalloc(&dinfo_array,batch);
	
	magma_malloc((void**)&darray_left, batch*sizeof(dt*));
	magma_malloc((void**)&darray_right, batch*sizeof(dt*));
	magma_malloc((void**)&dipiv_array, batch*sizeof(magma_int_t*));

	magma_ssetmatrix(r,r*batch,left,r,d_left,lddl,queue);
	magma_ssetmatrix(r,1*batch,right,r,d_right,lddr,queue);

	magma_sset_pointer(darray_left,d_left,lddl,0,0,lddl*r,batch,queue);
	magma_sset_pointer(darray_right,d_right,lddr,0,0,lddr,batch,queue);
	magma_iset_pointer(dipiv_array,d_ipiv,1,0,0,r,batch,queue);

	magma_sgesv_batched(
			row,
			nrhs,
			darray_left,
			lddl,
			dipiv_array,
			darray_right,
			lddr,
			dinfo_array,
			batch,
			queue
			);
	int *h_info = (int*)malloc(sizeof(int)*batch);
	magma_igetmatrix(batch,1,dinfo_array,batch,h_info,batch,queue);

//	for(int i = 0;i<batch;i++){
//		printf("%d ",h_info[i]);
//	}
//	printf("\n");
	magma_sgetmatrix(r,batch*1,d_right,lddr,res,r,queue);
	
//	printTensor(res,1,r*batch,1);
	
	free(h_info);
	magma_queue_destroy(queue);
	magma_free(d_left);
	magma_free(d_right);
	magma_free(darray_left);
	magma_free(darray_right);
	magma_free(d_ipiv);
	magma_free(dipiv_array);
	magma_free(dinfo_array);
	magma_finalize();

	dt *d_res = NULL;	
	dt *d_res1 = NULL;
	hipMalloc((void**)&d_res,sizeof(dt)*r*n);	//a*c	
	hipMalloc((void**)&d_res1,sizeof(dt)*r*n);	//b*c
	hipMemcpy(d_res,res,sizeof(dt)*n*r,hipMemcpyHostToDevice);
	dt alpha = 1.0;
	dt beta = 0.0;
	hipblasHandle_t handle1;
	hipblasCreate(&handle1);
	hipblasSgeam(
		handle1,
		HIPBLAS_OP_T,
		HIPBLAS_OP_N,
		n,
		r,
		&alpha,
		d_res,
		r,
		&beta,
		d_res1,
		n,
		d_res1,
		n
		 );
	hipMemcpy(res,d_res1,sizeof(dt)*n*r,hipMemcpyDeviceToHost);

	hipblasDestroy(handle1);
	hipFree(d_res);
	hipFree(d_res1);


}

