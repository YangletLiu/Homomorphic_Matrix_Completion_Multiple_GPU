#include "hip/hip_runtime.h"
/*************************************************************************
	> File Name: trust.cpp
	> Author: hanlu
	> Mail: hanlu@shu.edu.cn 
	> Created Time: 2018年12月10日 星期一 21时13分39秒
 ************************************************************************/

#include "head.h"

int main(int argc,char *argv[]){

	int M =atoi(argv[1]);
	int N = M;
	int R = int(0.01*M);
    
	int count_device = 0;
    	hipGetDeviceCount(&count_device);
    	if(count_device == 0){

    		printf("There is no GPUs\n");
	    	return false;
    	}
    	printf("There are %d GPUs\n",count_device);
    	for(int i=0;i<count_device;i++){
		hipDeviceProp_t prop;
	    	if(hipGetDeviceProperties(&prop,i) == hipSuccess){
			printf("GPU %d: \t %s \n",i,prop.name);
	    	}
    	}
	int gpu_0 = 0;
	int gpu_1 = 1;

	srand((unsigned)time(NULL));
	dt *A = new dt[M*R]();
	for(int i = 0;i<M*R;i++){
		A[i] = rand()*0.1/(RAND_MAX*0.1);
	}
	dt *B = new dt[N*R]();
	for(int i = 0;i<N*R;i++){
		B[i] = rand()*0.1/(RAND_MAX*0.1);
	}

	dt *D = new dt[M*N]();
	matrixProduct(A,B,D,M,R,N); // D is the origin data
//	cout<<"origin data"<<endl;
//	printTensor(D,1,M*N,1);   // we view it store as col
	delete[] A;A=nullptr;
	delete[] B;B=nullptr;



// get transpose of D  N*M
	dt *D1 = new dt[M*N]();
	for(int i = 0;i<M;i++){
		for(int j = 0;j<N;j++){
			D1[i*N+j] = D[j*M+i];
		}
	}
//	printTensor(D1,1,M*N,1);

//	dt Omega[M*N]={1,0,1,1,0,1,1,0,1,1,0,1};
	dt *Omega = new dt[M*N]();
	for(int i = 0;i<M*N;i++){
		Omega[i] = rand()*0.1/(RAND_MAX*0.1);
		if(Omega[i]<=0.1){
			Omega[i] = 1;
		}else{
			Omega[i] = 0;
		}
	}
	cout<<"sample is 0.5"<<endl;

//	printTensor(Omega,1,M*N,1);
	dt *Omega1 = new dt[M*N]();
	for(int i = 0;i<M;i++){
		for(int j = 0;j<N;j++){
			Omega1[i*N+j] = Omega[j*M+i];
		}
	}
//	printTensor(Omega1,1,M*N,1);

//	dt U[M*R] = {3,6,2,6,2,6,1,8};
	dt *U = new dt[M*R]();
	for(int i = 0;i<M*R;i++){
		U[i] = rand()*0.1/(RAND_MAX*0.1);
	}

	dt *V = new dt[R*N]();

	dt *left = new dt[R*R*N]();
	dt *right = new dt[R*1*N]();
	dt *left1 = new dt[R*R*M]();
	dt *right1 = new dt[R*1*M]();

	dt *PB = new dt[M*K]();   //public data size of M*R store as column
	for(int i = 0;i<M*K;i++){
		PB[i] = D[i];
	}
	
	dt *MM = new dt[M*N]();  //MM is encpypted matrix
	dt *MM1 = new dt[M*N]();  //MM1 is its transpose

	Enc(D,Omega,PB,MM,MM1,M,N);
/* MM is incom encryped matrix of size m*n
   Omega is sample matrix
   we divide n into two part and each solve on one gpu   
*/
	int size0 = N/2;
	int size1 = M/2;
    double start, end;
    start = omp_get_wtime();

	for(int i = 0;i<10;i++){
	
		hipSetDevice(gpu_0);
		getbatch(U,MM,Omega,left,right,M,size0,R);
		lsq(left,right,V,M,size0,R);

		hipSetDevice(gpu_1);
		getbatch(U,MM+M*size0,Omega+M*size0,left+R*R*size0,right+R*size0,M,N-size0,R);
		lsq(left+R*R*size0,right+R*size0,V+R*size0,M,N-size0,R);
		Matrixtranpose(V,N,R);
		hipDeviceSynchronize();
		
		hipSetDevice(gpu_0);
		getbatch(V,MM1,Omega1,left1,right1,N,size1,R);
		lsq(left1,right1,U,N,size1,R);

		hipSetDevice(gpu_1);
		getbatch(V,MM1+N*size1,Omega1+N*size1,left1+R*R*size1,right1+R*size1,N,M-size1,R);
		lsq(left1+R*R*size1,right1+R*size1,U+R*size1,N,M-size1,R);
		Matrixtranpose(U,M,R);
		hipDeviceSynchronize();

	}
	// we need to U*V
	dt *result1 = new dt[M*N]();
	mproduct(U,V,result1,M,R,N);
    end = omp_get_wtime();
    double time = end-start;
    cout<<time<<endl;
	dt *result = new dt[M*N]();
	Dec(result1,result,PB,M,N); // decrypt result1 to result

	delete[] left;left=nullptr;
	delete[] right;right=nullptr;
	delete[] left1;left1=nullptr;
	delete[] right1;right1=nullptr;
	delete[] Omega1;Omega1=nullptr;
	delete[] Omega;Omega=nullptr;
	delete[] D1;D1=nullptr;
	delete[] U;U=nullptr;
	delete[] V;V=nullptr;
//	printTensor(result,1,M*N,1);
	

	double sh = 0.0;
	double xia = 0.0;
	for(int i = 0;i<M*N;i++){
		sh+=(result[i]-D[i])*(result[i]-D[i]);
		xia+=(D[i]*D[i]);
	}
	delete[] result;result=nullptr;
	delete[] result1;result1=nullptr;
	delete[] D;D=nullptr;
	delete[] MM;MM=nullptr;
	delete[] MM1;MM1=nullptr;
	double error=0.0;
	error = sqrt(sh)/sqrt(xia);
	cout<<"error is  "<<error<<endl;

	return 0;
}

