/*************************************************************************
	> File Name: opera.h
	> Author: hanlu
	> Mail: hanlu@shu.edu.cn 
	> Created Time: 2018年12月10日 星期一 21时14分15秒
 ************************************************************************/

#include "head.h"

void printTensor(dt *A,int a,int b,int c){
	for(int i = 0;i<c;i++){
		for(int j = 0;j<a;j++){
			for(int k =0;k<b;k++){
				cout<<A[i*a*b+j*b+k]<<"  ";
			}
			cout<<endl;
		}
		cout<<"-----------------------------------"<<endl;
	}
	cout<<endl;
}

void matrixProduct(dt *A,dt *B,dt *C,int a,int b,int c){
	// A is a*b; B is b*c  C is a*c 
	dt *d_A;
	dt *d_B;
	hipMalloc((void**)&d_A,sizeof(dt)*a*b);
	hipMalloc((void**)&d_B,sizeof(dt)*b*c);
	dt *d_C;
	hipMalloc((void**)&d_C,sizeof(dt)*a*c);
	dt alpha = 1.0;
	dt beta = 0.0;
	hipMemcpy(d_A,A,sizeof(dt)*a*b,hipMemcpyHostToDevice);
	hipMemcpy(d_B,B,sizeof(dt)*b*c,hipMemcpyHostToDevice);

	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipblasSgemm(
			handle,
			HIPBLAS_OP_N,
			HIPBLAS_OP_N,
			a,
			c,
			b,
			&alpha,
			d_A,
			a,
			d_B,
			b,
			&beta,
			d_C,  
			a
			);
	hipMemcpy(C,d_C,sizeof(dt)*a*c,hipMemcpyDeviceToHost);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	hipblasDestroy(handle);
}


void mproduct(dt *A,dt *B,dt *C,int a,int b,int c){
	dt *d_A = NULL;	
	dt *d_B = NULL;	
	dt *d_C = NULL;
	hipMalloc((void**)&d_A,sizeof(dt)*a*b); 	//a*b	
	hipMalloc((void**)&d_B,sizeof(dt)*b*c);	//b*c	
	hipMalloc((void**)&d_C,sizeof(dt)*a*c);	//a*c
	hipMemcpy(d_A,A,sizeof(dt)*a*b,hipMemcpyHostToDevice);
	hipMemcpy(d_B,B,sizeof(dt)*b*c,hipMemcpyHostToDevice);
	dt alpha = 1.0;
	dt beta = 0.0;
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipblasSgemm(
		handle,
		HIPBLAS_OP_N,
		HIPBLAS_OP_T,
		a,
		c,
		b,
		&alpha,
		d_A,
		a,
		d_B,
		c,
		&beta,
		d_C,
		a
	    );

	hipMemcpy(C,d_C,sizeof(dt)*c*a,hipMemcpyDeviceToHost);
	hipblasDestroy(handle);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
}
