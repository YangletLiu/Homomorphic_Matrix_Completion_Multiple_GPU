#include "hip/hip_runtime.h"
#include "head.h"

__global__ void gettt(dt *A,dt *B,dt *C,int m,int r){
	
	int i = blockDim.x*blockIdx.x+threadIdx.x;
	const int temp = blockDim.x*gridDim.x;
	while(i<r*m){
		int row = i/m;
		int col = i%m;
		C[row*m+col] = B[col]*A[row*m+col];
		i+=temp;
	}
	__syncthreads();
}

void test(dt *A,dt *B,dt *C,int m,int r,int n){
	dt *d_A;
	hipMalloc((void**)&d_A,sizeof(dt)*m*r);
	hipMemcpy(d_A,A,sizeof(dt)*m*r,hipMemcpyHostToDevice);
		dt *t3 = new dt[m*r]();
		hipMemcpy(t3,d_A,sizeof(dt)*m*r,hipMemcpyDeviceToHost);
		cout<<"hhk"<<endl;
		printTensor(t3,r,m,1);
		delete[] t3;t3=nullptr;

	for(int i = 0;i<n;i++){
	
		dt *t2 = new dt[m*r]();
		hipMemcpy(t2,d_A,sizeof(dt)*m*r,hipMemcpyDeviceToHost);
		cout<<"kkk"<<endl;
		printTensor(t2,r,m,1);
		delete[] t2;t2=nullptr;

		dt *d_tt;
		hipMalloc((void**)&d_tt,sizeof(dt)*m*r);
		
		dt *d_sample;
		hipMalloc((void**)&d_sample,sizeof(dt)*m);
		hipMemcpy(d_sample,B+i*m,sizeof(dt)*m,hipMemcpyHostToDevice);
		dt *t1 = new dt[m]();
		hipMemcpy(t1,d_sample,sizeof(dt)*m,hipMemcpyDeviceToHost);
		printTensor(t1,m,1,1);
		delete[] t1;t1=nullptr;

		dim3 th(512,1,1);
		dim3 bl((m*r+512-1)/512,1,1);
		gettt<<<bl,th>>>(d_A,d_sample,d_tt,m,r);

		dt *tt = new dt[m*r]();
		hipMemcpy(tt,d_tt,sizeof(dt)*m*r,hipMemcpyDeviceToHost);
		printTensor(tt,r,m,1);
		delete[] tt;tt=nullptr;

		hipFree(d_tt);
		hipFree(d_sample);

	}
	hipFree(d_A);
}
