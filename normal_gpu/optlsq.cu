#include "head.h"

void base(dt *left,dt *res,dt *right,int zuo,int rank,int you){
	
	dt *d_left;
	hipMalloc((void**)&d_left,sizeof(dt)*zuo*rank);
	hipMemcpy(d_left,left,sizeof(dt)*zuo*rank,hipMemcpyHostToDevice);
	dt *d_right;
	hipMalloc((void**)&d_right,sizeof(dt)*zuo);
	hipMemcpy(d_right,right,sizeof(dt)*zuo,hipMemcpyHostToDevice);

	hipsolverHandle_t cusolverH = NULL;
	hipblasHandle_t cublasH = NULL;
	hipsolverDnCreate(&cusolverH);
	hipblasCreate(&cublasH);

//	int info_gpu=0;
	int *devInfo = NULL;
	hipMalloc((void**)&devInfo,sizeof(int));
	dt *d_work = NULL;
	int lwork = 0;
	dt *d_tau = NULL;
	hipMalloc((void**)&d_tau,sizeof(dt)*zuo);

	hipsolverDnSgeqrf_bufferSize(
			cusolverH,
			zuo,rank,
			d_left,zuo,
			&lwork
			);
	hipMalloc((void**)&d_work,sizeof(dt)*lwork);
	hipsolverDnSgeqrf(
			cusolverH,
			zuo,rank,
			d_left,zuo,
			d_tau,
			d_work,
			lwork,
			devInfo
			);
	hipDeviceSynchronize();
//	hipMemcpy(&info_gpu,devInfo,sizeof(int),hipMemcpyDeviceToHost);
//	cout<<info_gpu<<endl;

	//step2 compute Q^T*right

	hipsolverDnSormqr(
			cusolverH,
			HIPBLAS_SIDE_LEFT,
			HIPBLAS_OP_T,
			zuo,1,rank,
			d_left,zuo,
			d_tau,
			d_right,zuo,
			d_work,
			lwork,
			devInfo
			);
	hipDeviceSynchronize();
//	hipMemcpy(&info_gpu,devInfo,sizeof(int),hipMemcpyDeviceToHost);
//	cout<<info_gpu<<endl;

	// step3 solve R*x = Q^T*b 
	dt one = 1;
	hipblasStrsm(
			cublasH,
			HIPBLAS_SIDE_LEFT,
			HIPBLAS_FILL_MODE_UPPER,
			HIPBLAS_OP_N,
			HIPBLAS_DIAG_NON_UNIT,
			rank,1,
			&one,
			d_left,zuo,
			d_right,rank
			);
	hipDeviceSynchronize();
	hipMemcpy(res,d_right,sizeof(dt)*rank,hipMemcpyDeviceToHost);
	
	hipFree(d_tau);
	hipFree(d_work);
	hipFree(d_left);
	hipFree(d_right);
	hipFree(devInfo);
	hipblasDestroy(cublasH);
	hipsolverDnDestroy(cusolverH);
}
void lsq(dt *left,dt *res,dt *right,dt *sample,int zuo,int rank,int you){
	// right is m*n with unknown is set 0
	// left is m*r res is r*n 
	// sample 
	//step1 left = QR
	dt *d_left;
	hipMalloc((void**)&d_left,sizeof(dt)*zuo*rank);
	hipMemcpy(d_left,left,sizeof(dt)*zuo*rank,hipMemcpyHostToDevice);

	//initial jubin
	hipsolverHandle_t cusolverH = NULL;
	hipblasHandle_t cublasH = NULL;
	hipsolverDnCreate(&cusolverH);
	hipblasCreate(&cublasH);
	
for(int i = 0;i<you;i++){
	dt *d_temp;
	hipMalloc((void**)&d_temp,sizeof(dt)*zuo);
	hipMemcpy(d_temp,right+i*zuo,sizeof(dt)*zuo,hipMemcpyHostToDevice);

	dt *d_sample;
	hipMalloc((void**)&d_sample,sizeof(dt)*zuo);
	hipMemcpy(d_sample,sample+i*zuo,sizeof(dt)*zuo,hipMemcpyHostToDevice);

	dt *d_tt;
	hipMalloc((void**)&d_tt,sizeof(dt)*zuo*rank);

	dim3 thread(512,1,1);
	dim3 block((zuo*rank+512-1)/512,1,1);
	gettt<<<block,thread>>>(d_left,d_sample,d_tt,zuo,rank);
//	cout<<zuo<<"  "<<you<<endl;
/*	dt *t1 = new dt[zuo*rank]();
	hipMemcpy(t1,d_tt,sizeof(dt)*zuo*rank,hipMemcpyDeviceToHost);
	printTensor(t1,1,zuo*rank,1);
	delete[] t1;t1=nullptr;

	dt *t2 = new dt[zuo]();
	hipMemcpy(t2,d_temp,sizeof(dt)*zuo,hipMemcpyDeviceToHost);
	printTensor(t2,1,zuo,1);
	delete[] t2;t2=nullptr;
*/

//	int info_gpu = 0;
	int *devInfo = NULL;
	hipMalloc((void**)&devInfo,sizeof(int));
	dt *d_work = NULL;
	int lwork = 0;
	dt *d_tau = NULL;
	hipMalloc((void**)&d_tau,sizeof(dt)*zuo);

	hipsolverDnSgeqrf_bufferSize(
			cusolverH,
			zuo,rank,
			d_tt,zuo,
			&lwork
			);
	hipMalloc((void**)&d_work,sizeof(dt)*lwork);
	hipsolverDnSgeqrf(
			cusolverH,
			zuo,rank,
			d_tt,zuo,
			d_tau,
			d_work,
			lwork,
			devInfo
			);
	hipDeviceSynchronize();
//	hipMemcpy(&info_gpu,devInfo,sizeof(int),hipMemcpyDeviceToHost);
//	cout<<info_gpu<<endl;

	//step2 compute Q^T*right

	hipsolverDnSormqr(
			cusolverH,
			HIPBLAS_SIDE_LEFT,
			HIPBLAS_OP_T,
			zuo,1,rank,
			d_tt,zuo,
			d_tau,
			d_temp,zuo,
			d_work,
			lwork,
			devInfo
			);
	hipDeviceSynchronize();
//	hipMemcpy(&info_gpu,devInfo,sizeof(int),hipMemcpyDeviceToHost);
//	cout<<info_gpu<<endl;

	// step3 solve R*x = Q^T*b 
	dt one = 1;
	hipblasStrsm(
			cublasH,
			HIPBLAS_SIDE_LEFT,
			HIPBLAS_FILL_MODE_UPPER,
			HIPBLAS_OP_N,
			HIPBLAS_DIAG_NON_UNIT,
			rank,1,
			&one,
			d_tt,zuo,
			d_temp,rank
			);
	hipDeviceSynchronize();
	hipMemcpy(res+i*rank,d_temp,sizeof(dt)*rank,hipMemcpyDeviceToHost);
	
//	printTensor(res+i*rank,1,rank,1);
//	cout<<"ovber"<<endl;

	hipFree(d_tau);
	hipFree(d_work);
	hipFree(d_temp);
	hipFree(d_tt);
	hipFree(d_sample);
	hipFree(devInfo);

	
}
//	printTensor(res,n,r,1);
	// now res is n*r we want to transfer to r*n
	dt *d_res = NULL;	
	dt *d_res1 = NULL;
	hipMalloc((void**)&d_res,sizeof(dt)*you*rank);	//a*c	
	hipMalloc((void**)&d_res1,sizeof(dt)*you*rank);	//b*c
	hipMemcpy(d_res,res,sizeof(dt)*you*rank,hipMemcpyHostToDevice);
	dt alpha = 1.0;
	dt beta = 0.0;
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipblasSgeam(
		handle,
		HIPBLAS_OP_T,
		HIPBLAS_OP_N,
		you,
		rank,
		&alpha,
		d_res,
		rank,
		&beta,
		d_res1,
		you,
		d_res1,
		you
		 );
	hipMemcpy(res,d_res1,sizeof(dt)*you*rank,hipMemcpyDeviceToHost);
//	printTensor(res,1,n*r,1);

	hipblasDestroy(cublasH);
	hipsolverDnDestroy(cusolverH);
	hipFree(d_left);
	hipFree(d_res);
	hipFree(d_res1);
	hipDeviceReset();
}
