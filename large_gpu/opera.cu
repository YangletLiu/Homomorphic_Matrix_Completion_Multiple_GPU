/*************************************************************************
	> File Name: opera.h
	> Author: hanlu
	> Mail: hanlu@shu.edu.cn 
	> Created Time: 2018年12月10日 星期一 21时14分15秒
 ************************************************************************/

#include "head.h"

void printTensor(dt *A,int a,int b,int c){
	for(int i = 0;i<c;i++){
		for(int j = 0;j<a;j++){
			for(int k =0;k<b;k++){
				cout<<A[i*a*b+j*b+k]<<"  ";
			}
			cout<<endl;
		}
		cout<<"-----------------------------------"<<endl;
	}
	cout<<endl;
}

void newmode(dt *A,dt *B,dt *sample,dt *left,dt *right,int m,int n,int r){
	// A is a random initial m*r, B is m*n ,sample is m*n
	// the we need A mul each col of sample 
	// and B.*sample
	// left is r*r*n  right is r*1*n store the data we want
//	cout<<"3"<<endl;
	dt *Bsample = new dt[m*n]();
	for(int i = 0;i<m*n;i++){
		Bsample[i] = sample[i]*B[i];
	}  
//	cout<<"3"<<endl;
	dt *Asample = new dt[m*r*n]();
	for(int i = 0;i<n;i++){
		for(int j = 0;j<r;j++){
			for(int k = 0;k<m;k++){
				Asample[i*m*r+j*m+k]=A[j*m+k]*sample[i*m+k];
			}
		//	Asample[i*m*r+j] = A[j]*sample[j%m+i*m];
		}
	}
//	cout<<"3"<<endl;
//	printTensor(Bsample,1,m,n);
//	printTensor(Asample,1,m*r,n);

	// now we have got Asample m*r*n and Bsample m*1*n
	// then we sent each front slice to GPU to get we want
	for(int i = 0;i<n;i++){
		// transpose product self and store in col
		signalpro(Asample+i*m*r,left+i*r*r,m,r);
		signalpro1(Asample+i*m*r,Bsample+i*m,right+i*r,m,r);
	}
//	printTensor(left,1,r*r,n);
//	printTensor(right,1,r*1,n);

//	cout<<"3"<<endl;
	delete[] Asample;Asample = nullptr;
	delete[] Bsample;Bsample = nullptr;
	
}
void signalpro(dt *A,dt *C,int a,int b){
	// A is a*b 
	dt *d_A;
	hipMalloc((void**)&d_A,sizeof(dt)*a*b);
	dt *d_C;
	hipMalloc((void**)&d_C,sizeof(dt)*b*b);
	dt alpha = 1.0;
	dt beta = 0.0;
	hipMemcpy(d_A,A,sizeof(dt)*a*b,hipMemcpyHostToDevice);

	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipblasSgemm(
			handle,
			HIPBLAS_OP_T,
			HIPBLAS_OP_N,
			b,b,a,
			&alpha,
			d_A,a,
			d_A,a,
			&beta,
			d_C,b
			);
	hipMemcpy(C,d_C,sizeof(dt)*b*b,hipMemcpyDeviceToHost);
	hipFree(d_A);
	hipFree(d_C);
	hipblasDestroy(handle);
}
void signalpro1(dt *A,dt *B,dt *C,int a,int b){
	//compute A'*B 
	// A is a*b , B is a*1 ,C is b*1
	dt *d_A;
	dt *d_B;
	hipMalloc((void**)&d_A,sizeof(dt)*a*b);
	hipMalloc((void**)&d_B,sizeof(dt)*a*1);
	dt *d_C;
	hipMalloc((void**)&d_C,sizeof(dt)*b*1);
	dt alpha = 1.0;
	dt beta = 0.0;
	hipMemcpy(d_A,A,sizeof(dt)*a*b,hipMemcpyHostToDevice);
	hipMemcpy(d_B,B,sizeof(dt)*a*1,hipMemcpyHostToDevice);

	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipblasSgemm(
			handle,
			HIPBLAS_OP_T,
			HIPBLAS_OP_N,
			b,1,a,
			&alpha,
			d_A,a,
			d_B,a,
			&beta,
			d_C,b
			);
	hipMemcpy(C,d_C,sizeof(dt)*b*1,hipMemcpyDeviceToHost);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	hipblasDestroy(handle);

}
void matrixProduct(dt *A,dt *B,dt *C,int a,int b,int c){
	// A is a*b; B is b*c  C is a*c 
	dt *d_A;
	dt *d_B;
	hipMalloc((void**)&d_A,sizeof(dt)*a*b);
	hipMalloc((void**)&d_B,sizeof(dt)*b*c);
	dt *d_C;
	hipMalloc((void**)&d_C,sizeof(dt)*a*c);
	dt alpha = 1.0;
	dt beta = 0.0;
	hipMemcpy(d_A,A,sizeof(dt)*a*b,hipMemcpyHostToDevice);
	hipMemcpy(d_B,B,sizeof(dt)*b*c,hipMemcpyHostToDevice);

	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipblasSgemm(
			handle,
			HIPBLAS_OP_T,
			HIPBLAS_OP_T,
			a,
			c,
			b,
			&alpha,
			d_A,
			b,
			d_B,
			c,
			&beta,
			d_C,  
			a
			);
	hipMemcpy(C,d_C,sizeof(dt)*a*c,hipMemcpyDeviceToHost);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	hipblasDestroy(handle);
}


void mproduct(dt *A,dt *B,dt *C,int a,int b,int c){
	dt *d_A = NULL;	
	dt *d_B = NULL;	
	dt *d_C = NULL;
	hipMalloc((void**)&d_A,sizeof(dt)*a*b); 	//a*b	
	hipMalloc((void**)&d_B,sizeof(dt)*b*c);	//b*c	
	hipMalloc((void**)&d_C,sizeof(dt)*a*c);	//a*c
	hipMemcpy(d_A,A,sizeof(dt)*a*b,hipMemcpyHostToDevice);
	hipMemcpy(d_B,B,sizeof(dt)*b*c,hipMemcpyHostToDevice);
	dt alpha = 1.0;
	dt beta = 0.0;
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipblasSgemm(
		handle,
		HIPBLAS_OP_N,
		HIPBLAS_OP_T,
		a,
		c,
		b,
		&alpha,
		d_A,
		a,
		d_B,
		c,
		&beta,
		d_C,
		a
	    );

	hipMemcpy(C,d_C,sizeof(dt)*c*a,hipMemcpyDeviceToHost);
	hipblasDestroy(handle);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
}
