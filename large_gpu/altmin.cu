#include "head.h"

void lsq(dt *left,dt *right,dt *res,int m,int n,int r){
	//left is r*r*n res is r*1*n V is r*1*n
	// step1 point array replace old 
	hipsolverHandle_t handle = NULL;
	hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
	hipsolverDnCreate(&handle);

//	int *infoArray = new int[n]();
	int *d_infoArray = NULL;
	hipMalloc((void**)&d_infoArray,sizeof(int)*n);     //check flag

	dt **h_left = new dt*[n];
	dt **h_right = new dt*[n];

	// transfer data to GPU with pointarray
	for(int i = 0;i<n;i++){
		hipMalloc((void**)&h_left[i],sizeof(dt)*r*r);
		hipMemcpy(h_left[i],left+i*r*r,sizeof(dt)*r*r,hipMemcpyHostToDevice);
		hipMalloc((void**)&h_right[i],sizeof(dt)*r*1);		
		hipMemcpy(h_right[i],right+i*r*1,sizeof(dt)*r*1,hipMemcpyHostToDevice);
	}  
	hipDeviceSynchronize();

	dt **d_left; 
	dt **d_right;
	hipMalloc((void**)&d_left,sizeof(dt*)*n);
	hipMalloc((void**)&d_right,sizeof(dt*)*n);

	hipMemcpy(d_left,h_left,sizeof(dt*)*n,hipMemcpyHostToDevice);
	hipMemcpy(d_right,h_right,sizeof(dt*)*n,hipMemcpyHostToDevice);
	hipDeviceSynchronize();

	hipsolverDnSpotrfBatched(
		handle,
		uplo,
		r,
		d_left,r,
		d_infoArray,
		n
		);
	hipDeviceSynchronize();
/*	hipMemcpy(infoArray,d_infoArray,sizeof(int)*n,hipMemcpyDeviceToHost);
	for(int j = 0;j<n;j++){
		cout<<infoArray[j]<<"  ";
	}
	cout<<endl;
*/
	hipDeviceSynchronize();
	
	hipsolverDnSpotrsBatched(
		handle,
		uplo,
		r,1,
		d_left,r,
		d_right,r,
		d_infoArray,
		n
		);
	hipDeviceSynchronize();
/*	hipMemcpy(infoArray,d_infoArray,sizeof(int)*n,hipMemcpyDeviceToHost);
	for(int j = 0;j<n;j++){
		cout<<infoArray[j]<<"  ";
	}
	cout<<endl;
*/
	for(int i = 0;i<n;i++){
		hipMemcpy(res+i*r*1,h_right[i],sizeof(dt)*r*1,hipMemcpyDeviceToHost);
		printTensor(res+i*r*1,1,r,1);
	}
	hipDeviceSynchronize();

	hipFree(d_left);
	hipFree(d_right);
	hipFree(d_infoArray);
	delete[] h_left;h_left = nullptr;
	delete[] h_right;h_right = nullptr;
//	delete[] infoArray;infoArray=nullptr;

	//printTensor(res,n,r,1);
	// now res is n*r we want to transfer to r*n
	dt *d_res = NULL;	
	dt *d_res1 = NULL;
	hipMalloc((void**)&d_res,sizeof(dt)*r*n);	//a*c	
	hipMalloc((void**)&d_res1,sizeof(dt)*r*n);	//b*c
	hipMemcpy(d_res,res,sizeof(dt)*n*r,hipMemcpyHostToDevice);
	dt alpha = 1.0;
	dt beta = 0.0;
	hipblasHandle_t handle1;
	hipblasCreate(&handle1);
	hipblasSgeam(
		handle1,
		HIPBLAS_OP_T,
		HIPBLAS_OP_N,
		n,
		r,
		&alpha,
		d_res,
		r,
		&beta,
		d_res1,
		n,
		d_res1,
		n
		 );
	hipMemcpy(res,d_res1,sizeof(dt)*n*r,hipMemcpyDeviceToHost);

	hipblasDestroy(handle1);
	hipsolverDnDestroy(handle);
	hipFree(d_res);
	hipFree(d_res1);
	hipDeviceReset();

}

