#include "head.h"
void batchedtprod(float* t1,float* t2,float* T,hipblasOperation_t t_t1,hipblasOperation_t t_t2,int row, int col, int rank, int tupe) {
    int ht = tupe/2 + 1;//half tupe
    int bat1 = row*rank;
    int bat2 = col*rank;
    int bat = bat1 + bat2;
    float *d_t;
    //set stream
    hipStream_t stream[2];
    #pragma unroll
	for(int i=0;i<2;i++){
		hipStreamCreate(&stream[i]);
	}
    
    hipMalloc((void**)&d_t, tupe*bat*sizeof(float));
    hipfftComplex *d_fftData;
    hipMalloc((void**)&d_fftData,ht*bat*sizeof(hipfftComplex));
    hipMemcpyAsync(d_t,t1,tupe*bat1*sizeof(float),hipMemcpyHostToDevice,stream[0]);
    hipMemcpyAsync(d_t+tupe*bat1, t2,tupe*bat2*sizeof(float),hipMemcpyHostToDevice,stream[1]);
    //tfft
   
    hipfftHandle plan =0;
    hipfftHandle plan2 =0;


    int n[1] = {tupe};
    int stride = bat1, dist = 1;
    int in[1] = {tupe};
    int on[1] = {ht};
    hipfftSetStream(plan,stream[0]);
    hipfftSetStream(plan2,stream[1]);
    
    if(hipfftPlanMany(&plan,1, n, in, stride, dist, on, stride, dist,
                       HIPFFT_R2C, bat1)==HIPFFT_ALLOC_FAILED) {
            fprintf(stdout, "[%s]:[%d]CUFFT error: Plan creation failed",__FUNCTION__,__LINE__);
            return; 
        }
    if(hipfftExecR2C(plan, d_t,(hipfftComplex*)d_fftData)
         != HIPFFT_SUCCESS) {
            fprintf(stdout, "[%s]:[%d]CUFFT error: EXEC  failed",__FUNCTION__,__LINE__);
            return; 
        }
    
    int stride2 = bat2;
    if
        (hipfftPlanMany(&plan2,1, n, in, stride2, dist, on, stride2, dist,
                       HIPFFT_R2C, bat2)!=HIPFFT_SUCCESS) {
            fprintf(stdout, "[%s]:[%d]CUFFT error: Plan creation failed",__FUNCTION__,__LINE__);
            return; 
        }
    if
        (hipfftExecR2C(plan2, d_t+tupe*bat1,d_fftData+ht*bat1)
         != HIPFFT_SUCCESS) {
            fprintf(stdout, "[%s]:[%d]CUFFT error: EXEC  failed",__FUNCTION__,__LINE__);
            return; 
        }
	hipStreamSynchronize(stream[0]);
	hipStreamSynchronize(stream[1]);
	if(hipfftDestroy(plan)!=HIPFFT_SUCCESS){

		fprintf(stdout,"[%s]:[%d]cufftDestory failed!",__FUNCTION__,__LINE__);
		return;
	}
	if(hipfftDestroy(plan2)!=HIPFFT_SUCCESS){

		fprintf(stdout,"[%s]:[%d]cufftDestory failed!",__FUNCTION__,__LINE__);
		return;
	}
    hipFree(d_t);
	
	hipStreamDestroy(stream[0]);
	hipStreamDestroy(stream[1]);
    //gemmbatched

    hipfftComplex* d_Tf;
    hipMalloc((void**)&d_Tf,ht*row*col*sizeof(hipfftComplex));
    hipblasHandle_t handle;
    hipComplex alpha;
    alpha.x =1;
    alpha.y =0;
    hipComplex beta;
    beta.x = 0;
    beta.y = 0;
    int Am = row;
    int An = rank;
    int Bn = col;
    int Bm = rank;
    int lda = 0;
    int ldb = 0;
    int ldc = 0;
    int strA = Am*An;
    int strB = Bm*Bn;
    int strC = Am*Bn;
    if(t_t1==HIPBLAS_OP_N && t_t2==HIPBLAS_OP_N){
      lda = Am;
      ldb = Bm;
      ldc = Am;
    }else{
        if(t_t1==HIPBLAS_OP_N && t_t2==HIPBLAS_OP_C){
      lda = Am;
      ldb = Bn;
      ldc = Am;
        }else{
        if( t_t1==HIPBLAS_OP_C && t_t2==HIPBLAS_OP_N){
      lda = An;
      ldb = Bm;
      ldc = Am;
        }else{
        if(t_t1==HIPBLAS_OP_C && t_t2==HIPBLAS_OP_C){
      lda = An;
      ldb = Bn;
      ldc = Am;
        }else{
        printf("hipblasOperation_t error\n");
        }
        }
        }
    }
    hipblasCreate(&handle);
    hipblasCgemmStridedBatched(handle, t_t1, t_t2, Am, Bn, Bm,
            &alpha, d_fftData, lda, strA, d_fftData+Am*An*ht, ldb, strB, &beta,
            d_Tf, ldc, strC, ht);
    hipblasDestroy(handle);

    //Tifft

    hipfftHandle iplan;
    in[0] = ht;
    on[0] = tupe;
    float* d_T;
    bat = row*col;
    stride = bat;
    hipMalloc((void**)&d_T, tupe*bat*sizeof(float));
    if
        (hipfftPlanMany(&iplan,1, n, in, stride, dist, on, stride, dist,
                       HIPFFT_C2R, bat)==HIPFFT_INTERNAL_ERROR) {
            fprintf(stdout, "[%s]:[%d]CUIFFT error: Plan creation failed",__FUNCTION__,__LINE__);
            return; 
        }
    if
        (hipfftExecC2R(iplan,(hipfftComplex*)d_Tf, d_T)
         != HIPFFT_SUCCESS) {
            fprintf(stdout, "[%s]:[%d]CUIFFT error: EXEC  failed",__FUNCTION__,__LINE__);
            return; 
        }
    
    hipDeviceSynchronize();
    int threads=0;
    int blocks=0;
    int num=bat*tupe;
    if(tupe*bat<512){
       threads=num;
       blocks=1;
     }else{
	threads=512;
	blocks=(num%512 ==0)?num/512:num/512+1;
	}
    fftResultProcess<<<blocks,threads>>>(d_T,num,tupe);
    hipDeviceSynchronize();
    hipMemcpy(T,d_T,tupe*bat*sizeof(float),hipMemcpyDeviceToHost);

    if(hipfftDestroy(iplan)!=HIPFFT_SUCCESS){

	    fprintf(stdout,"[%s]:[%d]cufftDestory failed!",__FUNCTION__,__LINE__);
	    return;
    }
    hipFree(d_fftData);
    hipFree(d_Tf);
    hipFree(d_T);
//transform
    
    /*for (int i=0; i<tupe*bat; i++)
        T[i] = T[i]/tupe;*/

}
