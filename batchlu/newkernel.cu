#include "hip/hip_runtime.h"
#include "head.h"

/*__global__ void elepro(dt *t1,dt *t2,dt *tt,int k){
	
	int i = blockDim.x*blockIdx.x+threadIdx.x;
	const int temp = blockDim.x*gridDim.x;
	while(i<k){
		tt[i] = t1[i]*t2[i];
		i+=temp;
	}
	__syncthreads();

}

__global__ void elebat(dt *t1,dt *t2,dt *tt,int a,int b,int c){
// m r n
	int i = blockDim.x*blockIdx.x+threadIdx.x;
	const int temp = blockDim.x*gridDim.x;
	while(i<a*b*c){
		int tube = i/(a*b); //locate which slice
		int row = (i-tube*(a*b))/a;
		int col = (i-tube*(a*b))%a;
		tt[tube*a*b+row*a+col] = t1[row*a+col]*t2[col+tube*a];
		i+=temp;
	}
	__syncthreads();	
}
*/
__global__ void fftResultProcess(float* d_t,const int num,const int len){
const int tid = blockIdx.x*blockDim.x+threadIdx.x;
if(tid < num){
	d_t[tid]=d_t[tid]/len;
	}
	__syncthreads();
}

__global__ void fftResultProcess(hipComplex* d_t,const int num, const int len){
	const int tid = blockIdx.x*blockDim.x+threadIdx.x;
	if(tid < num){
	d_t[tid].x=d_t[tid].x/len;
	__syncthreads();
	d_t[tid].y=d_t[tid].y/len; 
	}
	__syncthreads();	
}

__global__ void elepro(dt *t1,dt *t2,dt *t3,dt *tt,int m,int n){
	
	int i = blockDim.x*blockIdx.x+threadIdx.x;
	const int temp = blockDim.x*gridDim.x;
	while(i<m*n){
		int col = i/m;
		int row = i%m;	
		tt[col*m+row] =((t1[col*m+row]*t2[col*m+row])*0.5+0.1*t3[row]+0.1*t3[m+row]+0.1*t3[2*m+row]+0.1*t3[3*m+row]+0.1*t3[4*m+row])*t2[col*m+row];
		i+=temp;
	}
	__syncthreads();

}
__global__ void elepro1(dt *t1,dt *t2,dt *tt,int m,int n){
	
	int i = blockDim.x*blockIdx.x+threadIdx.x;
	const int temp = blockDim.x*gridDim.x;
	while(i<m*n){
		int col = i/m;
		int row = i%m;	
		tt[col*m+row] =((t1[col*m+row])-0.1*t2[row]-0.1*t2[m+row]-0.1*t2[2*m+row]-0.1*t2[3*m+row]-0.1*t2[4*m+row])/0.5;
		i+=temp;
	}
	__syncthreads();
}
__global__ void obt(dt *X,dt *sample,dt *res,int m,int r,int n){
	
	int i = blockDim.x*blockIdx.x+threadIdx.x;  // number of threads
	int slice = i/1024;   // lock to compute which slice
	int tix = i%1024;
	__shared__ float tmp[4500];   //each block has share memory store one col of sample
	for(int j = 0;j<m;j++){
		tmp[j]=sample[slice*m+j];	
	}   //  now each block has their own shared memory

	while(tix<m*r){
		int row = tix%m;
		int col = tix/m;
		res[slice*m*r+col*m+row]=X[col*m+row]*tmp[row];

		tix=tix+1024;
	}	
}
void Dec(dt *A,dt *B,dt *PB,int m,int n){
	// completed matrix A, B is decrypted matrix 
	// PB is publlic data
	int k = K;
	dt *d_A;
	dt *d_B;
	dt *d_PB;
	hipMalloc((void**)&d_A,sizeof(dt)*m*n);
	hipMalloc((void**)&d_B,sizeof(dt)*m*n);
	hipMalloc((void**)&d_PB,sizeof(dt)*m*k);
	hipMemcpy(d_A,A,sizeof(dt)*m*n,hipMemcpyHostToDevice);
	hipMemcpy(d_PB,PB,sizeof(dt)*m*k,hipMemcpyHostToDevice);
	dim3 thread(512,1,1);
	dim3 block((m*n+512-1)/512,1,1);
	elepro1<<<block,thread>>>(d_A,d_PB,d_B,m,n);
	hipMemcpy(B,d_B,sizeof(dt)*m*n,hipMemcpyDeviceToHost);
	
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_PB);

}
void Enc(dt *D,dt *sample,dt *PB,dt *MM,dt *MM1,int m,int n){
	// D is origin matrix m*n sample is binary matrix m*n
	// PB is m*k is public data to encrypt
	// MM is encpypted matrix and MM1 is its transpose;
	int k = K;
	dt *d_D;
	dt *d_sample;
	dt *d_PB;

	dt *d_MM;
	hipMalloc((void**)&d_D,sizeof(dt)*m*n);
	hipMalloc((void**)&d_sample,sizeof(dt)*m*n);
	hipMalloc((void**)&d_PB,sizeof(dt)*m*k);

	hipMalloc((void**)&d_MM,sizeof(dt)*m*n);

	hipMemcpy(d_sample,sample,sizeof(dt)*m*n,hipMemcpyHostToDevice);
	hipMemcpy(d_D,D,sizeof(dt)*m*n,hipMemcpyHostToDevice);
	hipMemcpy(d_PB,PB,sizeof(dt)*m*k,hipMemcpyHostToDevice);
	dim3 thread(512,1,1);
	dim3 block((m*n+512-1)/512,1,1);
	elepro<<<block,thread>>>(d_D,d_sample,d_PB,d_MM,m,n);
	hipMemcpy(MM,d_MM,sizeof(dt)*m*n,hipMemcpyDeviceToHost);
     
	//m*n to n*m
	dt *d_MM1 = NULL;
	hipMalloc((void**)&d_MM1,sizeof(dt)*m*n);	//b*c
	dt alpha = 1.0;
	dt beta = 0.0;
	hipblasHandle_t handle1;
	hipblasCreate(&handle1);
	hipblasSgeam(
		handle1,
		HIPBLAS_OP_T,
		HIPBLAS_OP_N,
		m,
		n,
		&alpha,
		d_MM,
		n,
		&beta,
		d_MM1,
		m,
		d_MM1,
		m
		 );
	hipMemcpy(MM1,d_MM1,sizeof(dt)*m*n,hipMemcpyDeviceToHost);
	hipFree(d_MM);
	hipFree(d_MM1);
	hipFree(d_D);
	hipFree(d_sample);
	hipFree(d_PB);
	hipblasDestroy(handle1);

}

void getbatch(dt *A,dt *B,dt *sample,dt *left,dt *right,int m,int n,int r){
	// A is a random initial m*r, B is m*n, sample is m*n
	// the we need A mul each col of sample 
	// and B.*sample
	// left is r*r*n  right is r*1*n store the data we want
	dt *d_sample;
	dt *d_Bsample;
	hipMalloc((void**)&d_sample,sizeof(dt)*m*n);
	hipMalloc((void**)&d_Bsample,sizeof(dt)*m*n);
	hipMemcpy(d_sample,sample,sizeof(dt)*m*n,hipMemcpyHostToDevice);
	hipMemcpy(d_Bsample,B,sizeof(dt)*m*n,hipMemcpyHostToDevice);

/*	dt *temp = new dt[m*n]();
	hipMemcpy(temp,d_Bsample,sizeof(dt)*m*n,hipMemcpyDeviceToHost);
	printTensor(temp,1,m*n,1);
	delete[] temp;temp=nullptr;
*/
	dt *d_Asample;
	dt *d_A;
	hipMalloc((void**)&d_A,sizeof(dt)*m*r);
	hipMalloc((void**)&d_Asample,sizeof(dt)*m*r*n);
	hipMemcpy(d_A,A,sizeof(dt)*m*r,hipMemcpyHostToDevice);
	dim3 th2(1024,1,1);
	dim3 bl2(n,1,1);
	obt<<<bl2,th2>>>(d_A,d_sample,d_Asample,m,r,n);
	hipFree(d_A);
	hipFree(d_sample);

/*	dt *temp1 = new dt[m*r*n]();
	hipMemcpy(temp1,d_Asample,sizeof(dt)*m*r*n,hipMemcpyDeviceToHost);
	printTensor(temp1,1,m*r,n);
	delete[] temp1;temp1=nullptr;
*/
	// the we will compute batched product by hipblasSgemmStridedBatched
	// A'*Ax = A'*b
	// d_Asample is m*r*n d_Bsample is m*1*n

	dt *d_left;
	hipMalloc((void**)&d_left,sizeof(dt)*r*r*n);
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	dt alpha = 1.0;
	dt beta = 0.0;
	hipblasSgemmStridedBatched(
			handle,
			HIPBLAS_OP_T,
			HIPBLAS_OP_N,
			r,r,m,
			&alpha,
			d_Asample,m,m*r,
			d_Asample,m,m*r,
			&beta,
			d_left,r,r*r,
			n
			);
	
	hipMemcpy(left,d_left,sizeof(dt)*r*r*n,hipMemcpyDeviceToHost);
//	printTensor(left,1,r*r,n);
	
	dt *d_right;
	hipMalloc((void**)&d_right,sizeof(dt)*r*1*n);
	hipblasSgemmStridedBatched(
			handle,
			HIPBLAS_OP_T,
			HIPBLAS_OP_N,
			r,1,m,
			&alpha,
			d_Asample,m,m*r,
			d_Bsample,m,m,
			&beta,
			d_right,r,r,
			n
			);

	hipMemcpy(right,d_right,sizeof(dt)*r*1*n,hipMemcpyDeviceToHost);
//	printTensor(right,1,r*1,n);
	
	hipFree(d_Asample);
	hipFree(d_Bsample);
	hipFree(d_left);
	hipFree(d_right);

	hipblasDestroy(handle);

}
