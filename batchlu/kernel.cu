#include "hip/hip_runtime.h"
#include "head.h"

__global__ void fftResultProcess(float* d_t,const int num,const int len){
const int tid = blockIdx.x*blockDim.x+threadIdx.x;
if(tid < num){
	d_t[tid]=d_t[tid]/len;
	}
	__syncthreads();
}

__global__ void fftResultProcess(hipComplex* d_t,const int num, const int len){
	const int tid = blockIdx.x*blockDim.x+threadIdx.x;
	if(tid < num){
	d_t[tid].x=d_t[tid].x/len;
	__syncthreads();
	d_t[tid].y=d_t[tid].y/len; 
	}
	__syncthreads();	
}

__global__ void elepro(dt *t1,dt *t2,dt *tt,int k){
	
	int i = blockDim.x*blockIdx.x+threadIdx.x;
	const int temp = blockDim.x*gridDim.x;
	while(i<k){
		tt[i] = t1[i]*t2[i];
		i+=temp;
	}
	__syncthreads();

}

__global__ void elebat(dt *t1,dt *t2,dt *tt,int a,int b,int c){
// m r n
	int i = blockDim.x*blockIdx.x+threadIdx.x;
	const int temp = blockDim.x*gridDim.x;
	while(i<a*b*c){
		int tube = i/(a*b); //locate which slice
		int row = (i-tube*(a*b))/a;
		int col = (i-tube*(a*b))%a;
		tt[tube*a*b+row*a+col] = t1[row*a+col]*t2[col+tube*a];
		i+=temp;
	}
	__syncthreads();	
}
__global__ void obt(dt *X,dt *sample,dt *res,int m,int r,int n){
	
	int i = blockDim.x*blockIdx.x+threadIdx.x;  // number of threads
	int slice = i/1024;   // lock to compute which slice
	int tix = i%1024;
	__shared__ float tmp[4500];   //each block has share memory store one col of sample
	for(int j = 0;j<m;j++){
		tmp[j]=sample[slice*m+j];	
	}   //  now each block has their own shared memory

	while(tix<m*r){
		int row = tix%m;
		int col = tix/m;
		res[slice*m*r+col*m+row]=X[col*m+row]*tmp[row];

		tix=tix+1024;
	}
	
}
void getbatch(dt *A,dt *B,dt *sample,dt *left,dt *right,int m,int n,int r){
	// A is a random initial m*r, B is m*n, sample is m*n
	// the we need A mul each col of sample 
	// and B.*sample
	// left is r*r*n  right is r*1*n store the data we want
	dt *d_B;
	dt *d_sample;
	dt *d_Bsample;
	hipMalloc((void**)&d_B,sizeof(dt)*m*n);
	hipMalloc((void**)&d_sample,sizeof(dt)*m*n);
	hipMalloc((void**)&d_Bsample,sizeof(dt)*m*n);
	hipMemcpy(d_sample,sample,sizeof(dt)*m*n,hipMemcpyHostToDevice);
	hipMemcpy(d_B,B,sizeof(dt)*m*n,hipMemcpyHostToDevice);
	dim3 th(512,1,1);
	dim3 block1((m*n+512-1)/512,1,1);
	elepro<<<block1,th>>>(d_B,d_sample,d_Bsample,m*n);
	hipFree(d_B); // d_B is replace to d_Bsample

/*	dt *temp = new dt[m*n]();
	hipMemcpy(temp,d_Bsample,sizeof(dt)*m*n,hipMemcpyDeviceToHost);
	printTensor(temp,1,m*n,1);
	delete[] temp;temp=nullptr;
*/
	dt *d_Asample;
	dt *d_A;
	hipMalloc((void**)&d_A,sizeof(dt)*m*r);
	hipMalloc((void**)&d_Asample,sizeof(dt)*m*r*n);
	hipMemcpy(d_A,A,sizeof(dt)*m*r,hipMemcpyHostToDevice);
	dim3 block2((m*r*n+512-1)/512,1,1);
	elebat<<<block2,th>>>(d_A,d_sample,d_Asample,m,r,n);
	hipFree(d_A);
	hipFree(d_sample);

/*	dt *temp1 = new dt[m*r*n]();
	hipMemcpy(temp1,d_Asample,sizeof(dt)*m*r*n,hipMemcpyDeviceToHost);
	printTensor(temp1,1,m*r,n);
	delete[] temp1;temp1=nullptr;
*/
	// the we will compute batched product by hipblasSgemmStridedBatched
	// A'*Ax = A'*b
	// d_Asample is m*r*n d_Bsample is m*1*n

	dt *d_left;
	hipMalloc((void**)&d_left,sizeof(dt)*r*r*n);
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	dt alpha = 1.0;
	dt beta = 0.0;
	hipblasSgemmStridedBatched(
			handle,
			HIPBLAS_OP_T,
			HIPBLAS_OP_N,
			r,r,m,
			&alpha,
			d_Asample,m,m*r,
			d_Asample,m,m*r,
			&beta,
			d_left,r,r*r,
			n
			);
	
	hipMemcpy(left,d_left,sizeof(dt)*r*r*n,hipMemcpyDeviceToHost);
//	printTensor(left,1,r*r,n);
	
	dt *d_right;
	hipMalloc((void**)&d_right,sizeof(dt)*r*1*n);
	hipblasSgemmStridedBatched(
			handle,
			HIPBLAS_OP_T,
			HIPBLAS_OP_N,
			r,1,m,
			&alpha,
			d_Asample,m,m*r,
			d_Bsample,m,m,
			&beta,
			d_right,r,r,
			n
			);

	hipMemcpy(right,d_right,sizeof(dt)*r*1*n,hipMemcpyDeviceToHost);
//	printTensor(right,1,r*1,n);
	
	hipFree(d_Asample);
	hipFree(d_Bsample);
	hipFree(d_left);
	hipFree(d_right);

	hipblasDestroy(handle);

}
