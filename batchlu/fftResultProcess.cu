#include "hip/hip_runtime.h"
#include "based.h"
#if 1
__global__ void fftResultProcess(float* d_t,const int num,const int len){
const int tid = blockIdx.x*blockDim.x+threadIdx.x;
if(tid < num){
	d_t[tid]=d_t[tid]/len;
	}
	__syncthreads();
}

__global__ void fftResultProcess(hipComplex* d_t,const int num, const int len){
	const int tid = blockIdx.x*blockDim.x+threadIdx.x;
	if(tid < num){
	d_t[tid].x=d_t[tid].x/len;
	__syncthreads();
	d_t[tid].y=d_t[tid].y/len; 
	}
	__syncthreads();	
}

#endif

#if 0
__global__ void d_frp(float* d_t,const int num,const int len){
const int tid = blockIdx.x*blockDim.x+threadIdx.x;
if(tid < num){
	d_t[tid]=d_t[tid]/len;
	}
	__syncthreads();
}

__global__ void d_frp(hipComplex* d_t,const int num, const int len){
	const int tid = blockIdx.x*blockDim.x+threadIdx.x;
	if(tid < num){
	d_t[tid].x=d_t[tid].x/len; 
	d_t[tid].y=d_t[tid].y/len;
	}
	__syncthreads();	
}


void fffResultProcess(float* d_t,const int num,const int len){
	int threads = 0;
	int blocks = 0;
	if( num < 512){
	   threads = num;
	   blocks = 1;	
	}else{
	   threads = 512;
	   blocks = ((num%512) == 0)?num/512:num/512+1;	
	}
	d_frp<<<blocks,threads>>>(d_t,num,len);
}

void fffResultProcess(hipComplex* d_t,const int num,const int len){
	int threads = 0;
	int blocks = 0;
	if( num < 512){
	   threads = num;
	   blocks = 1;	
	}else{
	   threads = 512;
	   blocks = ((num%512) == 0)?num/512:num/512+1;	
	}
	d_frp<<<blocks,threads>>>(d_t,num,len);
}
#endif
