#include "head.h"

void base(dt *A,dt *B,dt *sample,dt *V,int m,int n,int r){
	

	dt *d_sample;
	dt *d_Bsample;
	hipMalloc((void**)&d_sample,sizeof(dt)*m*n);
	hipMalloc((void**)&d_Bsample,sizeof(dt)*m*n);
	hipMemcpy(d_sample,sample,sizeof(dt)*m*n,hipMemcpyHostToDevice);
	hipMemcpy(d_Bsample,B,sizeof(dt)*m*n,hipMemcpyHostToDevice);

	dt *d_Asample;
	dt *d_A;
	hipMalloc((void**)&d_A,sizeof(dt)*m*r);
	hipMalloc((void**)&d_Asample,sizeof(dt)*m*r*n);
	hipMemcpy(d_A,A,sizeof(dt)*m*r,hipMemcpyHostToDevice);
	dim3 th2(1024,1,1);
	dim3 bl2(n,1,1);
	obt<<<bl2,th2>>>(d_A,d_sample,d_Asample,m,r,n);
	hipFree(d_A);
	hipFree(d_sample);

	dt *d_left;
	hipMalloc((void**)&d_left,sizeof(dt)*r*r*n);
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	dt alpha = 1.0;
	dt beta = 0.0;
	hipblasSgemmStridedBatched(
			handle,
			HIPBLAS_OP_T,
			HIPBLAS_OP_N,
			r,r,m,
			&alpha,
			d_Asample,m,m*r,
			d_Asample,m,m*r,
			&beta,
			d_left,r,r*r,
			n
			);
	
	dt *d_right;
	hipMalloc((void**)&d_right,sizeof(dt)*r*1*n);
	hipblasSgemmStridedBatched(
			handle,
			HIPBLAS_OP_T,
			HIPBLAS_OP_N,
			r,1,m,
			&alpha,
			d_Asample,m,m*r,
			d_Bsample,m,m,
			&beta,
			d_right,r,r,
			n
			);
	// now left and right are in device memory
	// left is r*r*n right is r*1*n	

	hipsolverHandle_t cusolverH = NULL;
	hipblasHandle_t cublasH = NULL;
	hipsolverDnCreate(&cusolverH);
	hipblasCreate(&cublasH);

//	int info_gpu=0;
	int *devInfo = NULL;
	hipMalloc((void**)&devInfo,sizeof(int));
	dt *d_work = NULL;
	int lwork = 0;
	dt *d_tau = NULL;
	hipMalloc((void**)&d_tau,sizeof(dt)*r);

	for(int i = 0;i<n;i++){
	hipsolverDnSgeqrf_bufferSize(
			cusolverH,
			r,r,
			d_left+i*r*r,r,
			&lwork
			);
	hipMalloc((void**)&d_work,sizeof(dt)*lwork);
	hipsolverDnSgeqrf(
			cusolverH,
			r,r,
			d_left+i*r*r,r,
			d_tau,
			d_work,
			lwork,
			devInfo
			);
	hipDeviceSynchronize();
//	hipMemcpy(&info_gpu,devInfo,sizeof(int),hipMemcpyDeviceToHost);
//	cout<<info_gpu<<endl;

	//step2 compute Q^T*right

	hipsolverDnSormqr(
			cusolverH,
			HIPBLAS_SIDE_LEFT,
			HIPBLAS_OP_T,
			r,1,r,
			d_left+r*r*i,r,
			d_tau,
			d_right+i*r*1,r,
			d_work,
			lwork,
			devInfo
			);
	hipDeviceSynchronize();
//	hipMemcpy(&info_gpu,devInfo,sizeof(int),hipMemcpyDeviceToHost);
//	cout<<info_gpu<<endl;

	// step3 solve R*x = Q^T*b 
	dt one = 1;
	hipblasStrsm(
			cublasH,
			HIPBLAS_SIDE_LEFT,
			HIPBLAS_FILL_MODE_UPPER,
			HIPBLAS_OP_N,
			HIPBLAS_DIAG_NON_UNIT,
			r,1,
			&one,
			d_left+i*r*r,r,
			d_right+i*r*1,r
			);
	hipDeviceSynchronize();
	}
	hipMemcpy(V,d_right,sizeof(dt)*r*n,hipMemcpyDeviceToHost);
	
	hipFree(d_tau);
	hipFree(d_work);
	hipFree(d_left);
	hipFree(d_right);
	hipFree(devInfo);
	hipblasDestroy(cublasH);
	hipsolverDnDestroy(cusolverH);

	dt *d_V = NULL;	
	dt *d_res1 = NULL;
	hipMalloc((void**)&d_V,sizeof(dt)*r*n);	//a*c	
	hipMalloc((void**)&d_res1,sizeof(dt)*r*n);	//b*c
	hipMemcpy(d_V,V,sizeof(dt)*n*r,hipMemcpyHostToDevice);
	hipblasSgeam(
		handle,
		HIPBLAS_OP_T,
		HIPBLAS_OP_N,
		n,
		r,
		&alpha,
		d_V,
		r,
		&beta,
		d_res1,
		n,
		d_res1,
		n
		 );
	hipMemcpy(V,d_res1,sizeof(dt)*n*r,hipMemcpyDeviceToHost);

	hipblasDestroy(handle);
	hipFree(d_V);
	hipFree(d_res1);

}
